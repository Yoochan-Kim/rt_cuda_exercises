#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void helloKernel() {
    printf("Hello from GPU thread %d of block %d\n", threadIdx.x, blockIdx.x);
}

int main() {
    helloKernel<<<1, 4>>>();
    hipDeviceSynchronize();
    return 0;
}
