#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>
#include <numeric>
#include <random>

#define N (1024 * 1024 * 10)
#define BLOCK_SIZE 256
#define NUM_RUNS 20

inline void checkCuda(hipError_t e){ if(e!=hipSuccess){ fprintf(stderr,"CUDA error: %s\n",hipGetErrorString(e)); exit(1);} }

__global__ void k_coalesced(const int* __restrict__ in, int* __restrict__ out, int n){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n) out[idx] = in[idx] * 2;
}

__global__ void k_strided_read(const int* __restrict__ in, int* __restrict__ out, int n, int stride){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n){
        int j = (idx * stride) % n;
        out[idx] = in[j] * 2;
    }
}

__global__ void k_indirect_read(const int* __restrict__ in, const int* __restrict__ ind, int* __restrict__ out, int n){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n){
        int j = ind[idx];
        out[idx] = in[j] * 2;
    }
}

__global__ void k_trash(int* buf, int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n) buf[i] ^= 1;
}

static inline void trash_cache(int* d_dummy, int m){
    int grid = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    k_trash<<<grid, BLOCK_SIZE>>>(d_dummy, m);
    checkCuda(hipDeviceSynchronize());
}

static inline float run_coalesced(const int* d_in, int* d_out, int n){
    int grid = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    hipEvent_t s,t; checkCuda(hipEventCreate(&s)); checkCuda(hipEventCreate(&t));
    checkCuda(hipEventRecord(s));
    k_coalesced<<<grid, BLOCK_SIZE>>>(d_in, d_out, n);
    checkCuda(hipEventRecord(t));
    checkCuda(hipEventSynchronize(t));
    float ms=0; checkCuda(hipEventElapsedTime(&ms, s, t));
    checkCuda(hipEventDestroy(s)); checkCuda(hipEventDestroy(t));
    return ms;
}

static inline float run_strided(const int* d_in, int* d_out, int n, int stride){
    int grid = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    hipEvent_t s,t; checkCuda(hipEventCreate(&s)); checkCuda(hipEventCreate(&t));
    checkCuda(hipEventRecord(s));
    k_strided_read<<<grid, BLOCK_SIZE>>>(d_in, d_out, n, stride);
    checkCuda(hipEventRecord(t));
    checkCuda(hipEventSynchronize(t));
    float ms=0; checkCuda(hipEventElapsedTime(&ms, s, t));
    checkCuda(hipEventDestroy(s)); checkCuda(hipEventDestroy(t));
    return ms;
}

static inline float run_indirect(const int* d_in, const int* d_ind, int* d_out, int n){
    int grid = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    hipEvent_t s,t; checkCuda(hipEventCreate(&s)); checkCuda(hipEventCreate(&t));
    checkCuda(hipEventRecord(s));
    k_indirect_read<<<grid, BLOCK_SIZE>>>(d_in, d_ind, d_out, n);
    checkCuda(hipEventRecord(t));
    checkCuda(hipEventSynchronize(t));
    float ms=0; checkCuda(hipEventElapsedTime(&ms, s, t));
    checkCuda(hipEventDestroy(s)); checkCuda(hipEventDestroy(t));
    return ms;
}

int main(){
    const int n = N;
    const size_t bytes = (size_t)n * sizeof(int);

    std::vector<int> h_in(n);
    std::iota(h_in.begin(), h_in.end(), 0);

    std::vector<int> h_ind(n);
    std::iota(h_ind.begin(), h_ind.end(), 0);
    std::mt19937 rng(123);
    for(int i=n-1;i>0;--i){
        std::uniform_int_distribution<int> dist(0, i);
        int j = dist(rng);
        std::swap(h_ind[i], h_ind[j]);
    }

    int *d_in=nullptr, *d_out=nullptr, *d_ind=nullptr;
    checkCuda(hipMalloc(&d_in, bytes));
    checkCuda(hipMalloc(&d_out, bytes));
    checkCuda(hipMemcpy(d_in, h_in.data(), bytes, hipMemcpyHostToDevice));
    checkCuda(hipMalloc(&d_ind, bytes));
    checkCuda(hipMemcpy(d_ind, h_ind.data(), bytes, hipMemcpyHostToDevice));

    const int m = n * 2;
    int *d_dummy=nullptr;
    checkCuda(hipMalloc(&d_dummy, (size_t)m * sizeof(int)));
    checkCuda(hipMemset(d_dummy, 0, (size_t)m * sizeof(int)));

    int grid = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    k_coalesced<<<grid, BLOCK_SIZE>>>(d_in, d_out, n);
    checkCuda(hipDeviceSynchronize());
    trash_cache(d_dummy, m);

    float sum_coal=0, sum_stride=0, sum_rand=0;
    for(int r=0;r<NUM_RUNS;r++){
        trash_cache(d_dummy, m);
        sum_coal += run_coalesced(d_in, d_out, n);

        trash_cache(d_dummy, m);
        sum_stride += run_strided(d_in, d_out, n, 4);

        trash_cache(d_dummy, m);
        sum_rand += run_indirect(d_in, d_ind, d_out, n);
    }

    printf("Coalesced %.4f ms\n", sum_coal/NUM_RUNS);
    printf("Strided %.4f ms\n",   sum_stride/NUM_RUNS);
    printf("Random  %.4f ms\n",   sum_rand/NUM_RUNS);

    hipFree(d_dummy);
    hipFree(d_ind);
    hipFree(d_out);
    hipFree(d_in);
    return 0;
}
