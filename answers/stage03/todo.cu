#include "hip/hip_runtime.h"
// Stage 3 exercise: add two matrices on the GPU using 2D thread indexing.

#include <hip/hip_runtime.h>
#include "cuda_utils.cuh"
#include "matrix.h"

constexpr int kBlockDimX = 16;
constexpr int kBlockDimY = 16;

/* TODO:
 * Each thread should add the corresponding elements of a and b.
 * Steps:
 *   1) row = threadIdx.y; col = threadIdx.x;
 *   2) If row < height and col < width, store c[row * width + col] = a[...] + b[...].
 */
__global__ void addMatricesKernel(const float* a,
                                  const float* b,
                                  float* c,
                                  int width,
                                  int height) {
    const int row = threadIdx.y;
    const int col = threadIdx.x;
    if (row < height && col < width) {
        const int idx = row * width + col;
        c[idx] = a[idx] + b[idx];
    }
}

hipError_t addMatricesOnDevice(const Matrix& hostA,
                                const Matrix& hostB,
                                Matrix& hostC) {
    const std::size_t elementCount = matrixElementCount(hostA);
    const std::size_t byteSize = elementCount * sizeof(float);

    float* deviceA = nullptr;
    float* deviceB = nullptr;
    float* deviceC = nullptr;

    hipError_t status = hipMalloc(&deviceA, byteSize);
    if (status != hipSuccess) {
        return status;
    }
    status = hipMalloc(&deviceB, byteSize);
    if (status != hipSuccess) {
        hipFree(deviceA);
        return status;
    }
    status = hipMalloc(&deviceC, byteSize);
    if (status != hipSuccess) {
        hipFree(deviceB);
        hipFree(deviceA);
        return status;
    }

    status = hipMemcpy(deviceA, hostA.elements, byteSize, hipMemcpyHostToDevice);
    if (status != hipSuccess) {
        hipFree(deviceC);
        hipFree(deviceB);
        hipFree(deviceA);
        return status;
    }
    status = hipMemcpy(deviceB, hostB.elements, byteSize, hipMemcpyHostToDevice);
    if (status != hipSuccess) {
        hipFree(deviceC);
        hipFree(deviceB);
        hipFree(deviceA);
        return status;
    }

    /* TODO:
     * Configure the launch parameters and launch the kernel.
     * Hint: Use a single grid block (dim3 grid(1, 1)) and a 2D thread block
     *       whose dimensions are defined by kBlockDimX/kBlockDimY. Some threads
     *       will fall outside the matrix and rely on the boundary check.
     */
    const dim3 block(kBlockDimX, kBlockDimY);
    const dim3 grid(1, 1);
    addMatricesKernel<<<grid, block>>>(deviceA, deviceB, deviceC, hostA.width, hostA.height);
    status = hipGetLastError();
    if (status == hipSuccess) {
        status = hipDeviceSynchronize();
    }
    if (status != hipSuccess) {
        hipFree(deviceC);
        hipFree(deviceB);
        hipFree(deviceA);
        return status;
    }

    status = hipMemcpy(hostC.elements, deviceC, byteSize, hipMemcpyDeviceToHost);

    hipFree(deviceC);
    hipFree(deviceB);
    hipFree(deviceA);
    return status;
}
