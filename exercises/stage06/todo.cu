#include "hip/hip_runtime.h"
// Stage 6: Matrix Multiplication with Shared Memory
// Uses tiled algorithm with shared memory to reduce global memory bandwidth.

#include <hip/hip_runtime.h>
#include "cuda_utils.cuh"
#include "matrix_with_stride.h"

#define BLOCK_SIZE 16

// Get a matrix element
__device__ float GetElement(const Matrix A, int row, int col) {
    return A.elements[row * A.stride + col];
}

// Set a matrix element
__device__ void SetElement(Matrix A, int row, int col, float value) {
    A.elements[row * A.stride + col] = value;
}

// Get the BLOCK_SIZExBLOCK_SIZE sub-matrix Asub of A that is
// located col sub-matrices to the right and row sub-matrices down
// from the upper-left corner of A
__device__ Matrix GetSubMatrix(Matrix A, int row, int col) {
    Matrix Asub;
    Asub.width = BLOCK_SIZE;
    Asub.height = BLOCK_SIZE;
    Asub.stride = A.stride;
    Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row + BLOCK_SIZE * col];
    return Asub;
}

/* TODO:
 * Implement tiled matrix multiplication using shared memory.
 * Use GetSubMatrix to get tiles, load them into __shared__ arrays,
 * and synchronize with __syncthreads() before/after tile computation.
 */
__global__ void matrixMulKernel(Matrix A, Matrix B, Matrix C) {
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    Matrix Csub = GetSubMatrix(C, blockRow, blockCol);

    float Cvalue = 0.0f;

    int row = threadIdx.y;
    int col = threadIdx.x;

    for (int m = 0; m < (A.width / BLOCK_SIZE); ++m) {
        Matrix Asub = GetSubMatrix(A, blockRow, m);
        Matrix Bsub = GetSubMatrix(B, m, blockCol);

        // TODO: Declare shared memory for As and Bs tiles
        // TODO: Load tiles from global to shared memory
        // TODO: Synchronize before computation
        // TODO: Compute partial result and accumulate to Cvalue
        // TODO: Synchronize after computation
    }

    SetElement(Csub, row, col, Cvalue);
}

hipError_t multiplyMatricesOnDevice(const Matrix& hostA,
                                     const Matrix& hostB,
                                     Matrix& hostC) {
    // Load A and B to device memory
    Matrix deviceA;
    deviceA.width = deviceA.stride = hostA.width;
    deviceA.height = hostA.height;
    size_t size = hostA.width * hostA.height * sizeof(float);
    hipError_t status = hipMalloc(&deviceA.elements, size);
    if (status != hipSuccess) {
        return status;
    }
    status = hipMemcpy(deviceA.elements, hostA.elements, size, hipMemcpyHostToDevice);
    if (status != hipSuccess) {
        hipFree(deviceA.elements);
        return status;
    }

    Matrix deviceB;
    deviceB.width = deviceB.stride = hostB.width;
    deviceB.height = hostB.height;
    size = hostB.width * hostB.height * sizeof(float);
    status = hipMalloc(&deviceB.elements, size);
    if (status != hipSuccess) {
        hipFree(deviceA.elements);
        return status;
    }
    status = hipMemcpy(deviceB.elements, hostB.elements, size, hipMemcpyHostToDevice);
    if (status != hipSuccess) {
        hipFree(deviceB.elements);
        hipFree(deviceA.elements);
        return status;
    }

    // Allocate C in device memory
    Matrix deviceC;
    deviceC.width = deviceC.stride = hostC.width;
    deviceC.height = hostC.height;
    size = hostC.width * hostC.height * sizeof(float);
    status = hipMalloc(&deviceC.elements, size);
    if (status != hipSuccess) {
        hipFree(deviceB.elements);
        hipFree(deviceA.elements);
        return status;
    }

    /* TODO:
     * Launch the matrix multiplication kernel with a 2D grid.
     * Dimensions are multiples of BLOCK_SIZE, use simple division.
     */

    // Check for errors
    status = hipGetLastError();
    if (status == hipSuccess) {
        status = hipDeviceSynchronize();
    }
    if (status != hipSuccess) {
        hipFree(deviceC.elements);
        hipFree(deviceB.elements);
        hipFree(deviceA.elements);
        return status;
    }

    // Read C from device memory
    status = hipMemcpy(hostC.elements, deviceC.elements, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(deviceC.elements);
    hipFree(deviceB.elements);
    hipFree(deviceA.elements);

    return status;
}
