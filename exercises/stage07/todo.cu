#include "hip/hip_runtime.h"
// Stage 7: Shared Memory Reduction Baseline
// Implements the shared memory baseline using interleaved addressing.

#include <hip/hip_runtime.h>

#include <cstddef>
#include <vector>

#include "cuda_utils.cuh"

constexpr int kThreadsPerBlock = 1024;

/* TODO:
 * Implement the shared memory reduction using interleaved addressing.
 * Steps:
 *   1) Declare a shared memory buffer sized to blockDim.x (use extern __shared__).
 *   2) Load one element per thread from global memory if the global index is in range, otherwise store 0.
 *   3) For stride = 1, 2, 4, ... let only threads where threadIdx.x % (2 * stride) == 0 add their neighbor.
 *   4) After the loop, thread 0 writes the block's partial sum (sdata[0]) into g_odata[blockIdx.x].
 * Remember to keep __syncthreads() so that shared memory updates are visible before the next step.
 */
__global__ void reduceSharedMemoryKernel(const float* g_idata,
                                         float* g_odata,
                                         unsigned int count) {
    extern __shared__ float sdata[];

    const unsigned int tid = threadIdx.x;
    const unsigned int globalIdx = blockIdx.x * blockDim.x + tid;

    // TODO: Load g_idata[globalIdx] into shared memory (store 0.0f if the index is out of range).
    __syncthreads();

    for (unsigned int stride = 1; stride < blockDim.x; stride *= 2) {
        // TODO: Let only the threads that own multiples of 2*stride add their neighbor from shared memory.
        __syncthreads();
    }

    if (tid == 0) {
        // TODO: Write the block's partial sum (sdata[0]) to g_odata[blockIdx.x].
    }
}

// Host helper that prepares device buffers and collects per-block partial sums.
hipError_t reduceSharedMemoryBaseline(const float* hostInput,
                                       std::size_t count,
                                       float* outSum) {
    if (count == 0) {
        *outSum = 0.0f;
        return hipSuccess;
    }

    const std::size_t inputBytes = count * sizeof(float);
    float* deviceData = nullptr;
    hipError_t status = hipMalloc(&deviceData, inputBytes);
    if (status != hipSuccess) {
        return status;
    }

    status = hipMemcpy(deviceData, hostInput, inputBytes, hipMemcpyHostToDevice);
    if (status != hipSuccess) {
        hipFree(deviceData);
        return status;
    }

    const unsigned int threads = kThreadsPerBlock;
    const unsigned int gridSize =
        static_cast<unsigned int>((count + threads - 1) / threads);

    float* devicePartials = nullptr;
    status = hipMalloc(&devicePartials, gridSize * sizeof(float));
    if (status != hipSuccess) {
        hipFree(deviceData);
        return status;
    }

    std::vector<float> blockSums(gridSize, 0.0f);

    // Launch the shared memory baseline, synchronize, and copy the partial sums back into blockSums.
    const std::size_t sharedMemBytes = threads * sizeof(float);
    reduceSharedMemoryKernel<<<gridSize, threads, sharedMemBytes>>>(
        deviceData, devicePartials, static_cast<unsigned int>(count));

    status = hipGetLastError();
    if (status == hipSuccess) {
        status = hipDeviceSynchronize();
    }
    if (status == hipSuccess) {
        status = hipMemcpy(blockSums.data(),
                            devicePartials,
                            gridSize * sizeof(float),
                            hipMemcpyDeviceToHost);
    }

    const hipError_t kernelStatus = status;

    hipFree(devicePartials);
    hipFree(deviceData);

    if (kernelStatus != hipSuccess) {
        return kernelStatus;
    }

    double finalSum = 0.0;
    /* TODO:
     * Accumulate the partial sums using a double accumulator and store the final value in outSum.
     */
    // TODO: Reduce blockSums on the host and write the final total into outSum.

    *outSum = static_cast<float>(finalSum);
    return hipSuccess;
}
