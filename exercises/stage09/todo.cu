#include "hip/hip_runtime.h"
// Stage 9: Sequential Addressing Reduction
// Implements the sequential addressing variant by reversing the loop and using threadID-based indexing.

#include <hip/hip_runtime.h>

#include <cstddef>
#include <vector>

#include "cuda_utils.cuh"

constexpr int kThreadsPerBlock = 1024;

/* TODO:
 * Implement the shared memory reduction using sequential addressing to avoid bank conflicts.
 * Steps:
 *   1) Declare a shared memory buffer sized to blockDim.x (use extern __shared__).
 *   2) Load one element per thread from global memory if the global index is in range, otherwise store 0.
 *   3) For stride = blockDim.x / 2; stride > 0; stride >>= 1, let tid < stride accumulate sdata[tid + stride].
 *   4) After the loop, thread 0 writes the block's partial sum (sdata[0]) into g_odata[blockIdx.x].
 * Remember to keep __syncthreads() so that shared memory updates are visible before the next step.
 */
__global__ void reduceSharedMemoryKernel(const float* g_idata,
                                         float* g_odata,
                                         unsigned int count) {
    extern __shared__ float sdata[];

    const unsigned int tid = threadIdx.x;
    const unsigned int globalIdx = blockIdx.x * blockDim.x + tid;

    // TODO: Load g_idata[globalIdx] into shared memory (store 0.0f if the index is out of range).
    __syncthreads();

    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        // TODO: Let only the threads with tid < stride accumulate their neighbor from shared memory.
        __syncthreads();
    }

    if (tid == 0) {
        // TODO: Write the block's partial sum (sdata[0]) to g_odata[blockIdx.x].
    }
}

// Host helper that prepares device buffers and collects per-block partial sums.
hipError_t reduceSharedMemorySequentialAddressing(const float* hostInput,
                                                   std::size_t count,
                                                   float* outSum) {
    if (count == 0) {
        *outSum = 0.0f;
        return hipSuccess;
    }

    const std::size_t inputBytes = count * sizeof(float);
    float* deviceData = nullptr;
    hipError_t status = hipMalloc(&deviceData, inputBytes);
    if (status != hipSuccess) {
        return status;
    }

    status = hipMemcpy(deviceData, hostInput, inputBytes, hipMemcpyHostToDevice);
    if (status != hipSuccess) {
        hipFree(deviceData);
        return status;
    }

    const unsigned int threads = kThreadsPerBlock;
    const unsigned int gridSize =
        static_cast<unsigned int>((count + threads - 1) / threads);

    float* devicePartials = nullptr;
    status = hipMalloc(&devicePartials, gridSize * sizeof(float));
    if (status != hipSuccess) {
        hipFree(deviceData);
        return status;
    }

    std::vector<float> blockSums(gridSize, 0.0f);

    // Launch the sequential addressing reduction, synchronize, and copy the partial sums back into blockSums.
    const std::size_t sharedMemBytes = threads * sizeof(float);
    reduceSharedMemoryKernel<<<gridSize, threads, sharedMemBytes>>>(
        deviceData, devicePartials, static_cast<unsigned int>(count));
    status = hipGetLastError();
    if (status == hipSuccess) {
        status = hipDeviceSynchronize();
    }
    if (status == hipSuccess) {
        status = hipMemcpy(blockSums.data(),
                            devicePartials,
                            gridSize * sizeof(float),
                            hipMemcpyDeviceToHost);
    }

    const hipError_t kernelStatus = status;

    hipFree(devicePartials);
    hipFree(deviceData);

    if (kernelStatus != hipSuccess) {
        return kernelStatus;
    }

    double finalSum = 0.0;
    for (float partial : blockSums) {
        finalSum += static_cast<double>(partial);
    }

    *outSum = static_cast<float>(finalSum);
    return hipSuccess;
}
