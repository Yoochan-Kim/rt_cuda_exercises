// Stage 8: Early-Exit Warp Divergence
// Benchmarks divergent and stream-compaction variants for early-exit workloads.

#include <cstddef>
#include <vector>
#include <random>
#include <iomanip>
#include <iostream>

#include "cuda_utils.cuh"

#include "todo.cu"

constexpr std::size_t kElementCount = 1 << 24;  // 1M particles
constexpr int kBenchmarkRepeats = 100;

void generateParticles(std::vector<Particle>& particles) {
  std::mt19937 rng(12345u);
  std::uniform_real_distribution<float> energyDist(0.0f, 1.0f);
  std::uniform_real_distribution<float> posDist(-10.0f, 10.0f);
  std::uniform_real_distribution<float> velDist(-1.0f, 1.0f);

  for (Particle& p : particles) {
    p.energy = energyDist(rng);
  }
  for (Particle& p : particles) {
    p.x = posDist(rng);
  }
  for (Particle& p : particles) {
    p.y = posDist(rng);
  }
  for (Particle& p : particles) {
    p.z = posDist(rng);
  }
  for (Particle& p : particles) {
    p.vx = velDist(rng);
    p.vy = velDist(rng);
    p.vz = velDist(rng);
  }
}

int main() {
  std::vector<Particle> particles(kElementCount);
  generateParticles(particles);

  const std::size_t count = particles.size();
  const std::size_t particleBytes = count * sizeof(Particle);
  const std::size_t outputBytes = count * sizeof(StageValue);

  // Allocate device memory
  Particle* deviceInput = nullptr;
  StageValue* deviceOutput = nullptr;
  Particle* deviceCompacted = nullptr;
  int* deviceCount = nullptr;
  int* deviceIndices = nullptr;

  hipError_t status = hipMalloc(&deviceInput, particleBytes);
  if (status != hipSuccess) {
    std::cerr << "Stage 8 failed: " << hipGetErrorString(status) << std::endl;
    return 1;
  }

  status = hipMalloc(&deviceOutput, outputBytes);
  if (status != hipSuccess) {
    std::cerr << "Stage 8 failed: " << hipGetErrorString(status) << std::endl;
    hipFree(deviceInput);
    return 1;
  }

  status = hipMalloc(&deviceCompacted, particleBytes);
  if (status != hipSuccess) {
    std::cerr << "Stage 8 failed: " << hipGetErrorString(status) << std::endl;
    hipFree(deviceOutput);
    hipFree(deviceInput);
    return 1;
  }

  status = hipMalloc(&deviceCount, sizeof(int));
  if (status != hipSuccess) {
    std::cerr << "Stage 8 failed: " << hipGetErrorString(status) << std::endl;
    hipFree(deviceCompacted);
    hipFree(deviceOutput);
    hipFree(deviceInput);
    return 1;
  }

  status = hipMalloc(&deviceIndices, count * sizeof(int));
  if (status != hipSuccess) {
    std::cerr << "Stage 8 failed: " << hipGetErrorString(status) << std::endl;
    hipFree(deviceCount);
    hipFree(deviceCompacted);
    hipFree(deviceOutput);
    hipFree(deviceInput);
    return 1;
  }

  // Create events for timing
  hipEvent_t startEvent = nullptr;
  hipEvent_t stopEvent = nullptr;

  status = hipEventCreate(&startEvent);
  if (status != hipSuccess) {
    std::cerr << "Stage 8 failed: " << hipGetErrorString(status) << std::endl;
    hipFree(deviceIndices);
    hipFree(deviceCount);
    hipFree(deviceCompacted);
    hipFree(deviceOutput);
    hipFree(deviceInput);
    return 1;
  }

  status = hipEventCreate(&stopEvent);
  if (status != hipSuccess) {
    std::cerr << "Stage 8 failed: " << hipGetErrorString(status) << std::endl;
    hipEventDestroy(startEvent);
    hipFree(deviceIndices);
    hipFree(deviceCount);
    hipFree(deviceCompacted);
    hipFree(deviceOutput);
    hipFree(deviceInput);
    return 1;
  }

  // Copy input data
  status = hipMemcpy(deviceInput, particles.data(), particleBytes,
                      hipMemcpyHostToDevice);
  if (status != hipSuccess) {
    std::cerr << "Stage 8 failed: " << hipGetErrorString(status) << std::endl;
    hipEventDestroy(stopEvent);
    hipEventDestroy(startEvent);
    hipFree(deviceIndices);
    hipFree(deviceCount);
    hipFree(deviceCompacted);
    hipFree(deviceOutput);
    hipFree(deviceInput);
    return 1;
  }

  std::vector<StageValue> hostOutput(count);
  double divergentTime = 0.0;
  double compactionTime = 0.0;
  double divergentChecksum = 0.0;
  double compactionChecksum = 0.0;

  const char* failedLabel = nullptr;

  // Test 1: Divergent kernel with early-exit
  hipMemset(deviceOutput, 0, outputBytes);
  status = launchDivergentKernel(deviceInput, deviceOutput, count);
  if (status == hipSuccess) {
    status = hipDeviceSynchronize();
  }
  if (status == hipSuccess) {
    status = hipEventRecord(startEvent);
  }
  if (status == hipSuccess) {
    for (int iter = 0; iter < kBenchmarkRepeats; ++iter) {
      launchDivergentKernel(deviceInput, deviceOutput, count);
    }
    status = hipGetLastError();
  }
  if (status == hipSuccess) {
    status = hipEventRecord(stopEvent);
  }
  if (status == hipSuccess) {
    status = hipEventSynchronize(stopEvent);
  }
  if (status == hipSuccess) {
    float elapsedMs = 0.0f;
    status = hipEventElapsedTime(&elapsedMs, startEvent, stopEvent);
    if (status == hipSuccess) {
      divergentTime = static_cast<double>(elapsedMs) / kBenchmarkRepeats;
    }
  }
  if (status == hipSuccess) {
    status = hipMemcpy(hostOutput.data(), deviceOutput, outputBytes,
                        hipMemcpyDeviceToHost);
  }
  if (status == hipSuccess) {
    double sum = 0.0;
    for (StageValue v : hostOutput) {
      sum += static_cast<double>(v);
    }
    divergentChecksum = sum;
  }
  if (status != hipSuccess) {
    failedLabel = "divergent";
  }

  // Test 2: Stream compaction
  if (failedLabel == nullptr) {
    hipMemset(deviceOutput, 0, outputBytes);
    status = launchCompactionKernels(deviceInput, deviceCompacted, deviceCount,
                                      deviceIndices, deviceOutput, count);
    if (status == hipSuccess) {
      status = hipDeviceSynchronize();
    }
    if (status == hipSuccess) {
      status = hipEventRecord(startEvent);
    }
    if (status == hipSuccess) {
      for (int iter = 0; iter < kBenchmarkRepeats; ++iter) {
        launchCompactionKernels(deviceInput, deviceCompacted, deviceCount,
                                deviceIndices, deviceOutput, count);
      }
      status = hipGetLastError();
    }
    if (status == hipSuccess) {
      status = hipEventRecord(stopEvent);
    }
    if (status == hipSuccess) {
      status = hipEventSynchronize(stopEvent);
    }
    if (status == hipSuccess) {
      float elapsedMs = 0.0f;
      status = hipEventElapsedTime(&elapsedMs, startEvent, stopEvent);
      if (status == hipSuccess) {
        compactionTime = static_cast<double>(elapsedMs) / kBenchmarkRepeats;
      }
    }
    if (status == hipSuccess) {
      status = hipMemcpy(hostOutput.data(), deviceOutput, outputBytes,
                          hipMemcpyDeviceToHost);
    }
    if (status == hipSuccess) {
      double sum = 0.0;
      for (StageValue v : hostOutput) {
        sum += static_cast<double>(v);
      }
      compactionChecksum = sum;
    }
    if (status != hipSuccess) {
      failedLabel = "compaction";
    }
  }

  // Cleanup
  hipEventDestroy(stopEvent);
  hipEventDestroy(startEvent);
  hipFree(deviceIndices);
  hipFree(deviceCount);
  hipFree(deviceCompacted);
  hipFree(deviceOutput);
  hipFree(deviceInput);

  if (failedLabel != nullptr) {
    std::cerr << "Stage 8 " << failedLabel
              << " variant failed: " << hipGetErrorString(status) << std::endl;
    return 1;
  }

#ifdef SKIP_CPU
  std::cout << std::setprecision(10) << divergentChecksum << '\n'
            << compactionChecksum << std::endl;
  std::cerr << "Divergent time avg (ms): " << divergentTime << std::endl;
  std::cerr << "Compaction time avg (ms): " << compactionTime << std::endl;
#else
  std::cout << "Stage 8 early-exit divergence ✅" << std::endl;
  std::cout << "\nDataset:" << std::endl;
  std::cout << "  Particles: " << count << std::endl;
  std::cout << "  Block size: " << kThreadsPerBlock << std::endl;
  std::cout << "  Energy threshold: " << kEnergyThreshold << std::endl;
  std::cout << "  Compute iterations: " << kComputeIterations << std::endl;

  // Calculate active percentage
  int activeCount = 0;
  for (const Particle& p : particles) {
    if (p.energy >= kEnergyThreshold) {
      activeCount++;
    }
  }
  float activePercent = 100.0f * activeCount / count;

  std::cout << "  Active particles: " << activeCount << " (" << std::fixed
            << std::setprecision(1) << activePercent << "%)" << std::endl;

  std::cout << "\nResults (avg over " << kBenchmarkRepeats << " runs):"
            << std::endl;
  std::cout << std::fixed << std::setprecision(3);
  std::cout << "  Divergent (early-exit) : " << divergentTime << " ms | checksum "
            << std::setprecision(2) << divergentChecksum << std::setprecision(3)
            << std::endl;
  std::cout << "  Stream compaction      : " << compactionTime << " ms | checksum "
            << std::setprecision(2) << compactionChecksum << std::setprecision(3)
            << std::endl;

  std::cout << "\nSpeedup:" << std::endl;
  std::cout << std::setprecision(2);
  std::cout << "  Stream compaction : " << (divergentTime / compactionTime) << "x faster"
            << std::endl;
#endif

  return 0;
}
