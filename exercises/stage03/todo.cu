#include "hip/hip_runtime.h"
// Stage 3 exercise: add two matrices on the GPU using 2D thread indexing.

#include <hip/hip_runtime.h>
#include "cuda_utils.cuh"
#include "matrix.h"

constexpr int kBlockDimX = 16;
constexpr int kBlockDimY = 16;

/* TODO:
 * Each thread should add the corresponding elements of a and b.
 * Steps:
 *   1) row = threadIdx.y; col = threadIdx.x;
 *   2) If row < height and col < width, store c[row * width + col] = a[...] + b[...].
 */
__global__ void addMatricesKernel(const float* a,
                                  const float* b,
                                  float* c,
                                  int width,
                                  int height) {
    // TODO: implement this kernel
}

hipError_t addMatricesOnDevice(const Matrix& hostA,
                                const Matrix& hostB,
                                Matrix& hostC) {
    const std::size_t elementCount = matrixElementCount(hostA);
    const std::size_t byteSize = elementCount * sizeof(float);

    float* deviceA = nullptr;
    float* deviceB = nullptr;
    float* deviceC = nullptr;

    hipError_t status = hipMalloc(&deviceA, byteSize);
    if (status != hipSuccess) {
        return status;
    }
    status = hipMalloc(&deviceB, byteSize);
    if (status != hipSuccess) {
        hipFree(deviceA);
        return status;
    }
    status = hipMalloc(&deviceC, byteSize);
    if (status != hipSuccess) {
        hipFree(deviceB);
        hipFree(deviceA);
        return status;
    }

    status = hipMemcpy(deviceA, hostA.elements, byteSize, hipMemcpyHostToDevice);
    if (status != hipSuccess) {
        hipFree(deviceC);
        hipFree(deviceB);
        hipFree(deviceA);
        return status;
    }
    status = hipMemcpy(deviceB, hostB.elements, byteSize, hipMemcpyHostToDevice);
    if (status != hipSuccess) {
        hipFree(deviceC);
        hipFree(deviceB);
        hipFree(deviceA);
        return status;
    }

    /* TODO:
     * Configure the launch parameters and launch the kernel.
     * Hint: Use a single grid block (dim3 grid(1, 1)) and a 2D thread block
     *       whose dimensions are defined by kBlockDimX/kBlockDimY. Threads that
     *       fall outside the matrix bounds should be ignored by the kernel.
     */

    status = hipGetLastError();
    if (status == hipSuccess) {
        status = hipDeviceSynchronize();
    }
    if (status != hipSuccess) {
        hipFree(deviceC);
        hipFree(deviceB);
        hipFree(deviceA);
        return status;
    }

    status = hipMemcpy(hostC.elements, deviceC, byteSize, hipMemcpyDeviceToHost);

    hipFree(deviceC);
    hipFree(deviceB);
    hipFree(deviceA);
    return status;
}
